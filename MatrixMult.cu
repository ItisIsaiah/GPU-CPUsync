
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <time.h>



//initializes the matrix
void init_matrix(int *d, int N)
{
    for (int i = 0; i < N; i++)
    {
        d[i] = (rand() % 10);
    }
}


//optional Add matrix instead of multiply
__global__ void addMatrix(int *a, int *b, int *c)
{

    int global_index = threadIdx.x + blockDim.x * threadIdx.y;
    c[global_index] = a[global_index] + b[global_index];
}

//GPU matrix multiplication
__global__ void multMatrix(int *a,int *b,int *c,int width){
    int row= threadIdx.y + blockDim.y * blockIdx.y;
    int col= threadIdx.x + blockDim.x * blockIdx.x;
     
    
    if(row<width && col<width){
        for(int i=0; i<width;i++){
        c[row*width+col]+= a[row*width+i] * b[i*width+col];
        }       
    }
 
}

__global__ void kwait(unsigned long long duration){
    unsigned long long start=clock64();
    while(clock64()< start + duration);
}


//The benchmark for testing reasons
void testing(int N, int kernelLaunches, FILE *fptr){
    for (int i = 10; i <= N; i += 10)
    {

        clock_t start, end;
        double duration;
        srand(time(NULL));
        size_t bytes = i * i * sizeof(int);
        int *a, *b, *c;


        int rc=hipMallocManaged(&a, bytes);
        int rc2=hipMallocManaged(&b, bytes);
        int rc3=hipMallocManaged(&c,bytes);

        
      

        //Creates 16 threads per block and makes sure that there are 
        int threads =16;
        int blocks = (i + threads - 1) / threads;
        
        //in a 2D configuration to match the i*i space that we allocated previously
        dim3 THREADS(threads, threads);
        dim3 BLOCKS(blocks, blocks);


        //Initailizes the matrices with random numbers.
        init_matrix(a, i*i);
        init_matrix(b, i*i);

        unsigned long long my_duration = 20000000ULL;

        //timer for the benchmark and begins the benchmark
        start = clock();

        for (int j = 0; j <= kernelLaunches; j++)
        {
            kwait<<<1,1>>>(my_duration*i);
            //multMatrix<<<BLOCKS,THREADS>>>(a,b,c,i);
            hipDeviceSynchronize();
            //printf("Cuda Return Code: %d", rc);
            //printf("A=%d |B=%d| C=%d   |",a[i*i-1],b[i*i-1],c[i*i-1]);
        }
        end = clock();
        //ends the benchmark
        
    
        //Presents results
        duration = ((double)(end - start)) / CLOCKS_PER_SEC;

        
       // printf("Total Duration: %f \n", duration);
        printf("%f :", duration);
        
        double avgDuration = duration / kernelLaunches;
        
        //printf("Average time for each kernel:\n %f", avgDuration);
        printf(" %f :", avgDuration);

        //printf("Size %d\n", i);
        printf("%d\n", i);
        

        hipFree(a);
        hipFree(b);
        hipFree(c);
        
    }
}

int main(int argc, char **argv)
{
    
    FILE *fptr = NULL ;

    
   
    int N = 50;
    int kernelLaunches; 
    bool both=false;


    for (int i = 1; i < argc; i++) //Escape Values
    {
        if (strcmp(argv[i], "-size") == 0 && i + 1 < argc)
        {
            N = atoi(argv[i + 1]);
        }

        else if (strcmp(argv[i], "-n") == 0 && i+ 1 < argc)
        {
            kernelLaunches = atoi(argv[i + 1]);
        }
        else if (strcmp(argv[i], "-sync") == 0 && i + 1 < argc) {
            if (strcmp(argv[i + 1], "spin") == 0)
            {
                hipSetDeviceFlags(hipDeviceScheduleSpin);
               // fptr=fopen("spinResults.txt", "w");
               
            }
            else if (strcmp(argv[i + 1], "block") == 0)
            {
                //cudaSetDeviceFlags(cudaDeviceScheduleBlockingSync);
                hipSetDeviceFlags(hipDeviceScheduleAuto);
              //  fptr=fopen("blockResults.txt", "w");
               
            }
            else if (strcmp(argv[i + 1], "both") == 0)
            {
                both=true;
            }
            else
            {
                printf("\n INVALID SYNC");
            }
        }
    }

   // printf("I AM BREAKING HERE 177");
    
    
    //if you pciked a specific type of spin then just spin
    if(!both){
        //catches if the files doesnt exist
        
        testing(N,kernelLaunches,fptr);
       
    }
    //otherwise initialize and do both;
    else if(both){
        //printf("I AM BENCHMARKING BOTH");
        hipSetDeviceFlags(hipDeviceScheduleSpin);
        
        /*
        fptr=fopen("spinResults.txt", "w");
        //catches if the file doesnt exist
        
        if (fptr == NULL)
        {
            printf("Error opening file my g");
            exit(1);
        }
        */
        testing(N,kernelLaunches,fptr);
        


        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
        /*
        fptr=fopen("blockResults.txt", "w");
        
        //catches if the file doesnt exist
        if (fptr == NULL)
        {
            printf("Error opening file my g");
            exit(1);
        }
       */
        testing(N,kernelLaunches,fptr);

        //system("gnuplot -p blockResults.txt,spinResults.txt");
    }
}
