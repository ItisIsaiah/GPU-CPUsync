
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <time.h>
#include <iostream>

#include <sys/time.h>
#define USECPSEC 1000000ULL


//initializes the matrix
void init_matrix(int *d, int N)
{
    for (int i = 0; i < N; i++)
    {
        d[i] = (rand() % 10);
    }
}


//optional Add matrix instead of multiply
__global__ void addMatrix(int *a, int *b, int *c)
{

    int global_index = threadIdx.x + blockDim.x * threadIdx.y;
    c[global_index] = a[global_index] + b[global_index];
}

//GPU matrix multiplication
__global__ void multMatrix(int *a,int *b,int *c,int width){
    int row= threadIdx.y + blockDim.y * blockIdx.y;
    int col= threadIdx.x + blockDim.x * blockIdx.x;
     
    
    if(row<width && col<width){
        for(int i=0; i<width;i++){
        c[row*width+col]+= a[row*width+i] * b[i*width+col];
        }       
    }
 
}

unsigned long long dtime_usec(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
  //return ((tv.tv_sec)+tv.tv_usec)-start;
  //return ((tv.tv_sec*USECPSEC))-start;
}

__global__ void kwait(unsigned long long duration){
    unsigned long long start=clock64();
    while(clock64()< start + duration);
}


//The benchmark for testing reasons
void testing(int N, int kernelLaunches, FILE *fptr){
    for (int i = 5; i <= N; i += 5)
    {
        //printf("Seg fault is here 1");
        clock_t start, end;
        double duration;
        srand(time(NULL));
        size_t bytes = i * i * sizeof(int);
        int *a, *b, *c;


        int rc=hipMallocManaged(&a, bytes);
        int rc2=hipMallocManaged(&b, bytes);
        int rc3=hipMallocManaged(&c,bytes);

        
      

        //Creates 16 threads per block and makes sure that there are 
        int threads =16;
        int blocks = (i + threads - 1) / threads;
        
        //in a 2D configuration to match the i*i space that we allocated previously
        dim3 THREADS(threads, threads);
        dim3 BLOCKS(blocks, blocks);


        //Initailizes the matrices with random numbers.
        init_matrix(a, i*i);
        init_matrix(b, i*i);

        unsigned long long my_duration = 1000000ULL;

        //timer for the benchmark and begins the benchmark
       //start = clock();
        unsigned long long dt =dtime_usec(0);
       // unsigned long long gpuDT=dt+clock64();


        for (int j = 0; j <= kernelLaunches; j++)
        {
            //printf("Seg fault is here 7");
            kwait<<<1,1>>>((my_duration*1000)*(i/5));
           // multMatrix<<<BLOCKS,THREADS>>>(a,b,c,i);
            hipDeviceSynchronize();
            //printf("Cuda Return Code: %d", rc);
            //printf("A=%d |B=%d| C=%d   |",a[i*i-1],b[i*i-1],c[i*i-1]);
        }
        unsigned long long dtEND =dtime_usec(dt);
        //unsigned long long gpuEND=dtEND+clock64();
        //end = clock();
       
        unsigned long long expectedDuration=my_duration*(i/5);
        unsigned long long avgTimeTaken=dtEND/kernelLaunches;
        // divided by 1000 to get the number in NANOseconds
         //std::cout << "elapsed time: " << (dt/(float)USECPSEC)-(dtEND/(float)USECPSEC) << "s" << std::endl;
       // std::cout << expectedDuration << "|" << avgTimeTaken<< "|"<<expectedDuration-avgTimeTaken<<std::endl;
        std::cout << i <<"|"<<expectedDuration-avgTimeTaken<<std::endl;
        //ends the benchmark
        
        
        //Presents results
        //duration = dtEND/1000;

        /*
       // printf("Total Duration: %f \n", duration);
        printf("%f :", duration);
        
        double avgDuration = duration / kernelLaunches;
        
        //printf("Average time for each kernel:\n %f", avgDuration);
        printf(" %f :", avgDuration);

        //printf("Size %d\n", i);
        printf("%d\n", i);
        */

        hipFree(a);
        hipFree(b);
        hipFree(c);
        
    }
}

int main(int argc, char **argv)
{
    //printf("Seg fault is here 5");
    FILE *fptr = NULL ;

    
   
    int N = 50;
    int kernelLaunches; 
    bool both=false;


    for (int i = 1; i < argc; i++) //Escape Values
    {
        //printf("Seg fault is here");
        if (strcmp(argv[i], "-size") == 0 && i + 1 < argc)
        {
            N = atoi(argv[i + 1]);
        }

        else if (strcmp(argv[i], "-n") == 0 && i+ 1 < argc)
        {
            kernelLaunches = atoi(argv[i + 1]);
        }
        else if (strcmp(argv[i], "-sync") == 0 && i + 1 < argc) {
            if (strcmp(argv[i + 1], "spin") == 0)
            {
                hipSetDeviceFlags(hipDeviceScheduleSpin);    
            }
            else if (strcmp(argv[i + 1], "block") == 0)
            {
                hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
            }
            else if(strcmp(argv[i + 1], "auto") == 0){
                hipSetDeviceFlags(hipDeviceScheduleAuto);
            }
            else if(strcmp(argv[i + 1], "yield") == 0){
                hipSetDeviceFlags(hipDeviceScheduleYield);
            }
            else if(strcmp(argv[i + 1], "oldBlock") == 0){
                hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
            }
            else if (strcmp(argv[i + 1], "both") == 0)
            {
                both=true;
            }
            else
            {
                printf("\n INVALID SYNC");
            }
        }
    }

   // printf("I AM BREAKING HERE 177");
    
    
    //if you pciked a specific type of spin then just spin
    if(!both){
        //catches if the files doesnt exist
        
        testing(N,kernelLaunches,fptr);
       
    }
    //otherwise initialize and do both;
    else if(both){
        //printf("I AM BENCHMARKING BOTH");
        hipSetDeviceFlags(hipDeviceScheduleSpin);
        
        /*
        fptr=fopen("spinResults.txt", "w");
        //catches if the file doesnt exist
        
        if (fptr == NULL)
        {
            printf("Error opening file my g");
            exit(1);
        }
        */
        testing(N,kernelLaunches,fptr);
        


        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
        /*
        fptr=fopen("blockResults.txt", "w");
        
        //catches if the file doesnt exist
        if (fptr == NULL)
        {
            printf("Error opening file my g");
            exit(1);
        }
       */
        testing(N,kernelLaunches,fptr);

        //system("gnuplot -p blockResults.txt,spinResults.txt");
    }
}
